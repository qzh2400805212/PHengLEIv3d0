#include "cudaErrorHandle.h"

 //! handle error for cuda api
void handleAPIErr(hipError_t err, char const *file, const int line)
{
    if (err != hipSuccess)
    {
        cout << "Error: " << hipGetErrorString(err) << " in codes file " << file << " at line:# " << line << endl;
        exit(EXIT_FAILURE);
    }
}
 //! handle error for cuda kernel
void handleKernelErr(char const *file, const int line)
{
    hipError_t err;
    int         tline = line;  //! the handleKernelErr is called next to Kernel function
     //!err= hipGetLastError();
    hipDeviceSynchronize();
    err = hipPeekAtLastError();
    if (err != hipSuccess)
    {
        cout << "Error: Fail to Launch kernel in file " << file << " at line " << tline << endl
             << "Error Description: " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
}
